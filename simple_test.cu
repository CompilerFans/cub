#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>

// Include CUB headers
#include "cub/device/device_segmented_merge_sort.cuh"

int main()
{
    std::cout << "Starting DeviceSegmentedMergeSort simple test..." << std::endl;

    // Simple test data
    const int num_items = 8;
    const int num_segments = 2;
    
    // Host data
    std::vector<int> h_keys_in = {4, 2, 8, 1, 7, 3, 6, 5};  // Two segments: [4,2,8,1] and [7,3,6,5]
    std::vector<int> h_values_in = {0, 1, 2, 3, 4, 5, 6, 7};
    std::vector<int> h_segment_offsets = {0, 4, 8};  // First segment: 0-3, Second segment: 4-7
    
    std::vector<int> h_keys_out(num_items);
    std::vector<int> h_values_out(num_items);

    // Device pointers
    int *d_keys_in, *d_keys_out, *d_values_in, *d_values_out, *d_segment_offsets;
    
    // Allocate device memory
    hipMalloc(&d_keys_in, num_items * sizeof(int));
    hipMalloc(&d_keys_out, num_items * sizeof(int));
    hipMalloc(&d_values_in, num_items * sizeof(int));
    hipMalloc(&d_values_out, num_items * sizeof(int));
    hipMalloc(&d_segment_offsets, (num_segments + 1) * sizeof(int));
    
    // Copy data to device
    hipMemcpy(d_keys_in, h_keys_in.data(), num_items * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_values_in, h_values_in.data(), num_items * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_segment_offsets, h_segment_offsets.data(), (num_segments + 1) * sizeof(int), hipMemcpyHostToDevice);
    
    // Determine temp storage size
    void *d_temp_storage = nullptr;
    size_t temp_storage_bytes = 0;
    
    hipError_t result = hipcub::DeviceSegmentedMergeSort::SortPairs(
        d_temp_storage, temp_storage_bytes,
        d_keys_in, d_keys_out,
        d_values_in, d_values_out,
        num_items, num_segments,
        d_segment_offsets, d_segment_offsets + 1);
    
    std::cout << "Temp storage size: " << temp_storage_bytes << " bytes" << std::endl;
    
    if (result != hipSuccess) {
        std::cout << "Error determining temp storage: " << hipGetErrorString(result) << std::endl;
        return 1;
    }
    
    // Allocate temp storage
    hipMalloc(&d_temp_storage, temp_storage_bytes);
    
    // Run the sort
    result = hipcub::DeviceSegmentedMergeSort::SortPairs(
        d_temp_storage, temp_storage_bytes,
        d_keys_in, d_keys_out,
        d_values_in, d_values_out,
        num_items, num_segments,
        d_segment_offsets, d_segment_offsets + 1);
    
    if (result != hipSuccess) {
        std::cout << "Error running sort: " << hipGetErrorString(result) << std::endl;
        return 1;
    }
    
    // Copy results back
    hipMemcpy(h_keys_out.data(), d_keys_out, num_items * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(h_values_out.data(), d_values_out, num_items * sizeof(int), hipMemcpyDeviceToHost);
    
    // Print results
    std::cout << "Input keys:    ";
    for (int i = 0; i < num_items; i++) std::cout << h_keys_in[i] << " ";
    std::cout << std::endl;
    
    std::cout << "Output keys:   ";
    for (int i = 0; i < num_items; i++) std::cout << h_keys_out[i] << " ";
    std::cout << std::endl;
    
    std::cout << "Input values:  ";
    for (int i = 0; i < num_items; i++) std::cout << h_values_in[i] << " ";
    std::cout << std::endl;
    
    std::cout << "Output values: ";
    for (int i = 0; i < num_items; i++) std::cout << h_values_out[i] << " ";
    std::cout << std::endl;
    
    // Verify correctness (first segment should be [1,2,4,8], second segment should be [3,5,6,7])
    std::vector<int> expected_keys = {1, 2, 4, 8, 3, 5, 6, 7};
    bool correct = true;
    for (int i = 0; i < num_items; i++) {
        if (h_keys_out[i] != expected_keys[i]) {
            correct = false;
            break;
        }
    }
    
    std::cout << "Test result: " << (correct ? "PASSED" : "FAILED") << std::endl;
    
    // Cleanup
    hipFree(d_keys_in);
    hipFree(d_keys_out);
    hipFree(d_values_in);
    hipFree(d_values_out);
    hipFree(d_segment_offsets);
    hipFree(d_temp_storage);
    
    return correct ? 0 : 1;
}